#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <cmath>              // cos and sin

#define MATRIX_SIZE 4

// Function to create RX matrix
hipDoubleComplex* create_rx_matrix(double angle) {
    hipDoubleComplex* matrix = (hipDoubleComplex*)malloc(MATRIX_SIZE * sizeof(hipDoubleComplex));
    matrix[0] = (hipDoubleComplex){ cos(0.5 * angle), 0.0 };
    matrix[1] = (hipDoubleComplex){ 0.0, -sin(0.5 * angle) };
    matrix[2] = (hipDoubleComplex){ 0.0, -sin(0.5 * angle) };
    matrix[3] = (hipDoubleComplex){ cos(0.5 * angle), 0.0 };
    return matrix;
}

// Function to create RY matrix
hipDoubleComplex* create_ry_matrix(double angle) {
    hipDoubleComplex* matrix = (hipDoubleComplex*)malloc(MATRIX_SIZE * sizeof(hipDoubleComplex));
    matrix[0] = (hipDoubleComplex){ cos(0.5 * angle), 0.0 };
    matrix[1] = (hipDoubleComplex){ -sin(0.5 * angle), 0.0 };
    matrix[2] = (hipDoubleComplex){ sin(0.5 * angle), 0.0 };
    matrix[3] = (hipDoubleComplex){ cos(0.5 * angle), 0.0 };
    return matrix;
}

// Function to create RZ matrix
hipDoubleComplex* create_rz_matrix(double angle) {
    hipDoubleComplex* matrix = (hipDoubleComplex*)malloc(MATRIX_SIZE * sizeof(hipDoubleComplex));
    matrix[0] = (hipDoubleComplex){ cos(0.5 * angle), -sin(0.5 * angle) };
    matrix[1] = (hipDoubleComplex){ 0.0, 0.0 };
    matrix[2] = (hipDoubleComplex){ 0.0, 0.0 };
    matrix[3] = (hipDoubleComplex){ cos(0.5 * angle), sin(0.5 * angle) };
    return matrix;
}

int main(int argc, char** argv) {

    if (argc < 38) {
        printf("Usage: %s <depth> <angle1> <angle2> ... <angle36>\n", argv[0]);
        return EXIT_FAILURE;
    }

    int depth = atoi(argv[1]);

    // Store the 9 parameters from command line into a vector
    double angles[36];
    for (int i = 1; i < argc; ++i) {
        angles[i - 1] = atof(argv[i]);  // Convert command-line argument to double
    }
    
    const int nIndexBits = 3;                 // 3 qubits
    const int nSvSize    = (1 << nIndexBits); // 8 states 
    const int nTargets   = 1;                 //Always 1 target at a time
    const int nControls  = 1;                 //Always 1 control at a time
    const int adjoint    = 0;

    int32_t target0 = 0;
    int32_t target1 = 1;
    int32_t target2 = 2;

    int32_t control0 = 0;
    int32_t control1 = 1;

    // Create CUDA event for synchronization
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Define the initial state vector in host memory
    hipDoubleComplex h_sv[nSvSize] = {
        {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}
    }; // |000⟩

    hipDoubleComplex matrix_hadamard_2x2[] = {
        {1.0 / sqrt(2), 0.0},   {1.0 / sqrt(2), 0.0},
        {1.0 / sqrt(2), 0.0},  {-1.0 / sqrt(2), 0.0}
    };

    // Create matrices for each qubit
    hipDoubleComplex* rx_matrices[nIndexBits];
    hipDoubleComplex* ry_matrices[nIndexBits];
    hipDoubleComplex* rz_matrices[nIndexBits];

    hipDoubleComplex matrix_cnot[] = {{0.0, 0.0}, {1.0, 0.0},
                                     {1.0, 0.0}, {0.0, 0.0}};

    // Start recording time
    hipEventRecord(start);

    hipDoubleComplex* d_sv;
    hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex));

    hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex),
               hipMemcpyHostToDevice);

    //--------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    custatevecCreate(&handle);

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;
    //--------------------------------------------------------------------------

    // Apply Hadamard gates to the initial state
    for (int qubit = 0; qubit < 3; ++qubit) {
        int* target = (qubit == 0) ? &target0 : (qubit == 1) ? &target1 : &target2;
        custatevecApplyMatrix(
            handle, d_sv, HIP_C_64F, nIndexBits, matrix_hadamard_2x2, HIP_C_64F,
            CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, target, nTargets, nullptr,
            nullptr, 0, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes
        );
    }
    printf("\nState after Hadamard application:\n");
    hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
               hipMemcpyDeviceToHost);
    for (int i = 0; i < nSvSize; i++) {
        printf("(%f, %f)\t", h_sv[i].x, h_sv[i].y);
    }
    printf("\n");
    //--------------------------------------------------------------------------

    // Loop to apply each set of gates depth times
    for (int k = 0; k < depth; k++) {
            
        // Create matrices for each gate
        for (int j = 0; j < nIndexBits; ++j) {
            int baseIndex = k * 9;  // Base index in the angles array for each depth level
            rx_matrices[j] = create_rx_matrix(angles[baseIndex + j]);       // rx_matrices[j] corresponds to angles[k * 9 + j]
            ry_matrices[j] = create_ry_matrix(angles[baseIndex + nIndexBits + j]);  // ry_matrices[j] corresponds to angles[k * 9 + 3 + j]
            rz_matrices[j] = create_rz_matrix(angles[baseIndex + 2 * nIndexBits + j]);  // rz_matrices[j] corresponds to angles[k * 9 + 6 + j]
        }
        
        // Apply RX gates
        for (int j = 0; j < nIndexBits; ++j) {
            int32_t* target = (j == 0) ? &target0 : (j == 1) ? &target1 : &target2;
            custatevecApplyMatrix(
                handle, d_sv, HIP_C_64F, nIndexBits, rx_matrices[j], HIP_C_64F,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, target, nTargets, nullptr,
                nullptr, 0, CUSTATEVEC_COMPUTE_64F,
                extraWorkspace, extraWorkspaceSizeInBytes
            );
        }

        // Apply RY gates
        for (int j = 0; j < nIndexBits; ++j) {
            int32_t* target = (j == 0) ? &target0 : (j == 1) ? &target1 : &target2;
            custatevecApplyMatrix(
                handle, d_sv, HIP_C_64F, nIndexBits, ry_matrices[j], HIP_C_64F,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, target, nTargets, nullptr,
                nullptr, 0, CUSTATEVEC_COMPUTE_64F,
                extraWorkspace, extraWorkspaceSizeInBytes
            );
        }

        // Apply RZ gates
        for (int j = 0; j < nIndexBits; ++j) {
            int32_t* target = (j == 0) ? &target0 : (j == 1) ? &target1 : &target2;
            custatevecApplyMatrix(
                handle, d_sv, HIP_C_64F, nIndexBits, rz_matrices[j], HIP_C_64F,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, target, nTargets, nullptr,
                nullptr, 0, CUSTATEVEC_COMPUTE_64F,
                extraWorkspace, extraWorkspaceSizeInBytes
            );
        }

        // Apply the CNOT gates
        custatevecApplyMatrix(
            handle, d_sv, HIP_C_64F, nIndexBits, matrix_cnot, HIP_C_64F,
            CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, &target1, nTargets, &control0,
            nullptr, nControls, CUSTATEVEC_COMPUTE_64F,
            extraWorkspace, extraWorkspaceSizeInBytes
        );
        custatevecApplyMatrix(
            handle, d_sv, HIP_C_64F, nIndexBits, matrix_cnot, HIP_C_64F,
            CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, &target2, nTargets, &control1,
            nullptr, nControls, CUSTATEVEC_COMPUTE_64F,
            extraWorkspace, extraWorkspaceSizeInBytes
        );  

    // Print the state after each set of operations
    printf("\nState after iteration %d:\n", k + 1);
    hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
            hipMemcpyDeviceToHost);
    for (int i = 0; i < nSvSize; i++) {
        printf("(%f, %f)\t", h_sv[i].x, h_sv[i].y);
    }
    printf("\n");
    }

    //--------------------------------------------------------------------------

    // Stop recording time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //--------------------------------------------------------------------------

    // destroy events
    custatevecDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //--------------------------------------------------------------------------

    // Copy result back to host memory
    hipDoubleComplex h_sv_result[nSvSize];
    hipMemcpy(h_sv_result, d_sv, nSvSize * sizeof(hipDoubleComplex),
               hipMemcpyDeviceToHost);

    hipFree(d_sv);
    // Free allocated memory
    for (int i = 0; i < nIndexBits; ++i) {
        free(rx_matrices[i]);
    }

    // Calculate and print elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for quantum operations: %f ms\n", milliseconds);

    //Do the measurement
    float h_measures[nSvSize];
    for (int i = 0; i < nSvSize; i++){
        h_measures[i] = h_sv_result[i].x * h_sv_result[i].x  + h_sv_result[i].y * h_sv_result[i].y;
    }
    
    printf("Final State:\n");
    for (int i = 0; i < nSvSize; i++) {
        printf("(%f, %f)\t", h_sv_result[i].x, h_sv_result[i].y);
    }
    printf("\n");
    for (int i = 0; i < nSvSize; i++) {
        printf("measurement %d: %f\n", i, h_measures[i]);
    }

    return EXIT_SUCCESS;
}