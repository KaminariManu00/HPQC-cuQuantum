#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main() {
    
    const int nIndexBits = 2;  // 2 qubits
    const int nSvSize    = (1 << nIndexBits); // 4 states: |00>, |01>, |10>, |11>
    const int nTargets   = 2;
    const int adjoint    = 0;

    int targets[]  = {0,1};

    // Define the initial state vector in host memory
    hipDoubleComplex h_sv[] = {
        {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}
    }; // |00⟩

    hipDoubleComplex matrix_hadamard[] = {
        {1.0 / 2.0, 0.0},   {1.0 / 2.0, 0.0},   {1.0 / 2.0, 0.0},   {1.0 / 2.0, 0.0},
        {1.0 / 2.0, 0.0},   {-1.0 / 2.0, 0.0},  {1.0 / 2.0, 0.0},   {-1.0 / 2.0, 0.0},
        {1.0 / 2.0, 0.0},   {1.0 / 2.0, 0.0},   {-1.0 / 2.0, 0.0},  {-1.0 / 2.0, 0.0},
        {1.0 / 2.0, 0.0},   {-1.0 / 2.0, 0.0},  {-1.0 / 2.0, 0.0},  {1.0 / 2.0, 0.0}
    };

    hipDoubleComplex matrix_hadamard_2x2[] = {
        {1.0 / sqrt(2), 0.0},   {1.0 / sqrt(2), 0.0},
        {1.0 / sqrt(2), 0.0},  {-1.0 / sqrt(2), 0.0}
    };

    hipDoubleComplex* d_sv;

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event using CUDA event
    hipEventRecord(start);

    hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex));

    hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex),
               hipMemcpyHostToDevice);

    //--------------------------------------------------------------------------

    

    // custatevec handle initialization
    custatevecHandle_t handle;
    custatevecCreate(&handle);

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    //--------------------------------------------------------------------------
    // Apply Hadamard gates to the initial state |00>
    custatevecApplyMatrix(
        handle, d_sv, HIP_C_64F, nIndexBits, matrix_hadamard, HIP_C_64F,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, nullptr,
        nullptr, 0, CUSTATEVEC_COMPUTE_64F,
        extraWorkspace, extraWorkspaceSizeInBytes);

    //--------------------------------------------------------------------------

    // Apply Hadamard gates again to the output to measure
    custatevecApplyMatrix(
        handle, d_sv, HIP_C_64F, nIndexBits, matrix_hadamard, HIP_C_64F,
        CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, nullptr,
        nullptr, 0, CUSTATEVEC_COMPUTE_64F,
        extraWorkspace, extraWorkspaceSizeInBytes);
    //--------------------------------------------------------------------------

    // destroy handle
    custatevecDestroy(handle);

    //--------------------------------------------------------------------------

    // Copy result back to host memory
    hipDoubleComplex h_sv_result[nSvSize];
    hipMemcpy(h_sv_result, d_sv, nSvSize * sizeof(hipDoubleComplex),
               hipMemcpyDeviceToHost);

    hipFree(d_sv);

    hipEventRecord(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total Time elapsed: %f ms\n", milliseconds);

    // Check if the result is correct after applying Hadamard
    bool correct = true;
    for (int i = 0; i < nSvSize; i++) {
        if ((h_sv_result[i].x != h_sv[i].x) ||
            (h_sv_result[i].y != h_sv[i].y)) {
            correct = false;
            break;
        }
    }

    if (correct)
        printf("Hadamard gates test PASSED\n");
    else
        printf("Hadamard gates test FAILED: wrong result\n");

    return EXIT_SUCCESS;
}